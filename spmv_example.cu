#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>

int main() {
    // Exemplo: matriz 3x3 esparsa em CSR
    int rows = 3, cols = 3, nnz = 4;

    // CSR representation (host)
    int h_csrRowPtr[] = {0, 1, 3, 4};      // tamanho = n_rows + 1
    int h_csrColInd[] = {0, 0, 2, 1};      // tamanho = nnz
    float h_csrVal[]  = {10, 20, 30, 40};  // tamanho = nnz

    float h_x[] = {1, 2, 3};               // vetor de entrada
    float h_y[3];                          // resultado

    // Alocar memória na GPU
    int *d_csrRowPtr, *d_csrColInd;
    float *d_csrVal, *d_x, *d_y;

    hipMalloc((void**)&d_csrRowPtr, (rows + 1) * sizeof(int));
    hipMalloc((void**)&d_csrColInd, nnz * sizeof(int));
    hipMalloc((void**)&d_csrVal, nnz * sizeof(float));
    hipMalloc((void**)&d_x, cols * sizeof(float));
    hipMalloc((void**)&d_y, rows * sizeof(float));

    // Copiar dados para GPU
    hipMemcpy(d_csrRowPtr, h_csrRowPtr, (rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd, h_csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, h_csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, cols * sizeof(float), hipMemcpyHostToDevice);

    // Criar handle cuSPARSE
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Criar descritor da matriz esparsa
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreateCsr(&matA, rows, cols, nnz,
                      d_csrRowPtr, d_csrColInd, d_csrVal,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // Vetores densos
    hipsparseCreateDnVec(&vecX, cols, d_x, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, rows, d_y, HIP_R_32F);

    // Buffer temporário
    size_t bufferSize;
    void* dBuffer = nullptr;
    float alpha = 1.0f, beta = 0.0f;

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);

    hipMalloc(&dBuffer, bufferSize);

    // Executar SpMV
    hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

    // Copiar resultado para CPU
    hipMemcpy(h_y, d_y, rows * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Resultado y = A*x:\n";
    for (int i = 0; i < rows; ++i) std::cout << h_y[i] << " ";
    std::cout << "\n";

    // Liberar memória e handles
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    hipFree(d_csrRowPtr); hipFree(d_csrColInd); hipFree(d_csrVal);
    hipFree(d_x); hipFree(d_y); hipFree(dBuffer);

    return 0;
}
