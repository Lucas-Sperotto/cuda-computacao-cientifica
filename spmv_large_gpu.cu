#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>

// Parâmetros da matriz
const int N = 20000;          // Matriz N x N
const int NNZ = 2000000;      // 2 milhões de elementos não-nulos (~0.5%)

int main() {
    // Gerador aleatório
    std::mt19937 rng(12345);
    std::uniform_int_distribution<int> col_dist(0, N - 1);
    std::uniform_real_distribution<float> val_dist(-1.0f, 1.0f);

    // Alocar estruturas CSR na CPU
    std::vector<int> h_csrRowPtr(N + 1, 0);
    std::vector<int> h_csrColInd;
    std::vector<float> h_csrVal;

    int nnz_per_row = NNZ / N;

    for (int i = 0; i < N; ++i) {
        h_csrRowPtr[i + 1] = h_csrRowPtr[i] + nnz_per_row;
        for (int j = 0; j < nnz_per_row; ++j) {
            h_csrColInd.push_back(col_dist(rng));
            h_csrVal.push_back(val_dist(rng));
        }
    }

    // Vetor x e y
    std::vector<float> h_x(N, 1.0f);
    std::vector<float> h_y(N, 0.0f);

    // Alocar GPU
    int *d_csrRowPtr, *d_csrColInd;
    float *d_csrVal, *d_x, *d_y, *dBuffer;
    hipMalloc(&d_csrRowPtr, (N + 1) * sizeof(int));
    hipMalloc(&d_csrColInd, NNZ * sizeof(int));
    hipMalloc(&d_csrVal, NNZ * sizeof(float));
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Copiar dados para GPU
    hipMemcpy(d_csrRowPtr, h_csrRowPtr.data(), (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd, h_csrColInd.data(), NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, h_csrVal.data(), NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x.data(), N * sizeof(float), hipMemcpyHostToDevice);

    // Criar handle cuSPARSE
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Criar descritores
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreateCsr(&matA, N, N, NNZ, d_csrRowPtr, d_csrColInd, d_csrVal,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&vecX, N, d_x, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, N, d_y, HIP_R_32F);

    // Buffer temporário
    size_t bufferSize;
    float alpha = 1.0f, beta = 0.0f;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Cronometrar execução massiva
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 100000; ++i) {
        hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
            HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Tempo para 100000 SpMV grandes: " << elapsed.count() << " segundos\n";

    // Limpeza
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    hipFree(d_csrRowPtr); hipFree(d_csrColInd); hipFree(d_csrVal);
    hipFree(d_x); hipFree(d_y); hipFree(dBuffer);

    return 0;
}